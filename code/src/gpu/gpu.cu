#include "solver.h"

#include "solve.cuh"

/*Copyright (C) 2025

  This program is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>.

*/

#define ERR_DEVCNT "Couldn't check CUDA compatibility.\n"
#define ERR_NOCUDA "This machine has no CUDA capable devices.\n"
#define WARN_EXTRAGPUS "Warning, this machine has multiple CUDA devices.\n" \
                       "    Only device 0 will be used.\n"

global_variable hipDeviceProp_t DeviceProperties;

/**
 * Returns the version of the solver.
 */
u64
solver_Version()
{
    return 2;
}

/**
 * Returns the version of the main program the solver is compatible with.
 */
u64
solver_Compatibility()
{
    return 2;
}

/**
 * Returns the name of the solver.
 */
const char *
solver_Name()
{
    return "GPU Parallel Genetic Solver";
}

/**
 * Returns the description of the solver.
 */
const char *
solver_Description()
{
    return "Closely follows the elite island model from the paper at "
        "http://difu100cia.uaz.edu.mx/index.php/difuciencia/article/view/145";
}

/**
 * Checks if there is a CUDA device and loads its data.
 *
 * @return 1 (everything ok) or 0 (error).
 */
b32
solver_Setup()
{
    i32 DeviceCount = 0;
    hipError_t Err = hipGetDeviceCount(&DeviceCount);

    if (Err != hipSuccess)
    {
        IGNORE_RESULT(write(1, ERR_DEVCNT, sizeof(ERR_DEVCNT)));
        return 0;
    }

    if (DeviceCount == 0)
    {
        IGNORE_RESULT(write(1, ERR_NOCUDA, sizeof(ERR_NOCUDA)));
        return 0;
    }
    else if (DeviceCount > 1)
    {
        IGNORE_RESULT(write(1, WARN_EXTRAGPUS, sizeof(WARN_EXTRAGPUS)));
        return 0;
    }

    hipSetDevice(0);
    hipGetDeviceProperties(&DeviceProperties, 0);

    return 1;
}

/**
 * Does any necessary library unloading procedures.
 */
void
solver_Unload() {}

/**
 * Executes the euclidean TSP solver.
 *
 * @param Tsp A pointer to the TSP instance to be read from.
 * @param out_Permutation An array to be filled in with the best solution.
 * @param Iterations Has the maximum number of iterations and the ones executed
 *                   are written back into it.
 * @param Cutoff If this value is reached or passed, we stop immediately.
 * @param Parallelism Number of islands (thread blocks), if zero lets the
 *                    solver decide.
 * 
 * @return 1 if everything is ok, or 0 otherwise.
 */
b32
solver_Solve(tsp_instance *__restrict__ Tsp,
             i32 *__restrict__ out_Permutation,
             u64 *__restrict__ Iterations,
             r32 Cutoff,
             i32 Parallelism)
{
    return Main(Tsp, out_Permutation, Iterations, &DeviceProperties, Cutoff, Parallelism);
}
